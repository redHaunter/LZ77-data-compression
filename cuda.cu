﻿#include "hip/hip_runtime.h"


#include <iostream>
#include <string>
#include <vector>
#include <fstream>
#include <sstream>
#include <chrono>
#include <cassert>
#include <iostream>
#include <algorithm>
#include <iomanip>
#include <cstring>

using namespace std;
using namespace std::chrono;

struct token
{
    int offset;
    int length;
    char next;
};

__global__ void lz77_compress_kernel(char *input, int input_length, token **output, int window_size, int buffer_size, int num_theards, int string_length)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int start_index = input_length * idx;
    int i = 0;
    int end = input_length;
    if (idx == num_theards - 1)
    {
        end += string_length - (input_length * num_theards);
    }
    int index_output = 0;
    while (i < end)
    {
        int j = max(0, i - window_size);
        int longest_match_length = 0;
        int best_match_offset = 0;

        while (j < i)
        {
            int k = 0;
            while (i + k < end && input[j + k + start_index] == input[i + k + start_index] && k < buffer_size)
            {
                k++;
            }
            if (k > longest_match_length)
            {
                longest_match_length = k;
                best_match_offset = i - j;
            }

            j++;
        }
        if (longest_match_length > 0)
        {
            output[idx][index_output++] = {best_match_offset, longest_match_length, input[i + longest_match_length + start_index]};
            i += longest_match_length + 1;
        }
        else
        {
            output[idx][index_output++] = {0, 0, input[i + start_index]};
            i++;
        }
    }
}

vector<vector<token>> lz77_compress_cuda(string input, int window_size, int buffer_size, int num_blocks, int num_theards)
{
    char *input_dev;
    token **output_dev;
    int input_length = input.length();

    hipMalloc((void **)&input_dev, input_length * sizeof(char));
    hipMalloc((void **)&output_dev, num_theards * num_blocks * sizeof(token *));

    hipMemcpy(input_dev, input.c_str(), input_length * sizeof(char), hipMemcpyHostToDevice);

    token **output = new token *[num_theards * num_blocks];
    for (int i = 0; i < num_theards * num_blocks; i++)
    {
        hipMalloc((void **)&output[i], input_length * sizeof(token));
    }
    hipMemcpy(output_dev, output, num_theards * num_blocks * sizeof(token *), hipMemcpyHostToDevice);
    int t = input_length / (num_theards * num_blocks);
    vector<vector<token>> output_vectors(num_theards * num_blocks);
    lz77_compress_kernel<<<num_blocks, num_theards>>>(input_dev, t, output_dev, window_size, buffer_size, num_theards * num_blocks, input_length);
    for (int i = 0; i < num_theards * num_blocks; i++)
    {
        output_vectors[i].resize(input_length);
        hipMemcpy(&(output_vectors[i][0]), output[i], input_length * sizeof(token), hipMemcpyDeviceToHost);
        hipFree(output[i]);
    }
    hipFree(input_dev);
    hipFree(output_dev);
    delete[] output;

    return output_vectors;
}

string lz77_decompress(vector<vector<token>> compressed)
{
    string output = "";
    for (auto block : compressed)
    {
        for (auto t : block)
        {
            if (t.length == 0)
            {
                output += t.next;
            }
            else
            {
                int start = output.length() - t.offset;
                for (int i = 0; i < t.length; i++)
                {
                    output += output[start + i];
                }
                output += t.next;
            }
        }
    }
    return output;
}

int main()
{
    ofstream timeResult("Time result2.csv");
    ofstream sizeResult("Size result2.csv");
    timeResult << "2,4,8,16,32,64,128,256,512\n";
    sizeResult << "2,4,8,16,32,64,128,256,512\n";
    for (int i = 2; i <= 32; i *= 2)
    {
        for (int j = 2; j <= 32; j *= 2)
        {
            string filename = "C:\\Users\\Asus\\Desktop\\cuda_cpp\\8.txt";
            ifstream infile(filename);
            string input((istreambuf_iterator<char>(infile)), istreambuf_iterator<char>());
            cout << "Size of actual file: " << input.size() << endl;
            int window_size = 100;
            int buffer_size = 10;
            auto start = high_resolution_clock::now();
            vector<vector<token>> compressed = lz77_compress_cuda(input, window_size, buffer_size, i, j);
            auto stop = high_resolution_clock::now();

            int sum = 0;
            for (vector<token> v : compressed)
            {
                for (token vv : v)
                {
                    if (vv.next == '\0')
                        break;
                    sum++;
                }
            }
            cout << "Size of compress file: " << sum << endl;
            auto duration = duration_cast<microseconds>(stop - start);
            cout << "Time taken " << duration.count() << " microseconds" << endl;
            sizeResult << sum << ",";
            timeResult << duration.count() << ",";
            hipDeviceReset();
        }
        sizeResult << endl;
        timeResult << endl;
    }

    timeResult.close();
    sizeResult.close();

    // string decompressed = lz77_decompress(compressed);
    // cout << decompressed << endl;
    return 0;
}
